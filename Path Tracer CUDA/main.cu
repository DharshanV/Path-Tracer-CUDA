#include <iostream>
#include "GLFW.h"
#include "Renderer.h"
#include "VertexArray.h"
#include "ElementBuffer.h"
#include "Shader.h"
#include "Texture.h"

using namespace std;
using namespace glm;

void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void mouse_callback(GLFWwindow* window, double xpos, double ypos);
void updateTexture(Renderer& renderer, hipArray_t* writeTo);
void processInput(GLFWwindow* window);
void createScene(Renderer& renderer);
bool lockFPS(uint32_t FPS);

float lastX = WIDTH / 2.0f;
float lastY = HEIGHT / 2.0f;
bool firstMouse = true;

Camera camera;
int samples = SAMPLE_MIN;

float deltaTime = 0.0f;
float lastFrame = 0.0f;
bool globalLight = true;

int main() {
	srand(time(0));
	//==================
	//Setup GLFW and OpenGL
	GLFW window(WIDTH, HEIGHT, "Path Tracer");

	if (!window.isGood()) { glfwTerminate(); return EXIT_FAILURE; }
	window.setClearColor({ 0,0,0,1.0f });

	glfwSetInputMode(window.getWindow(), GLFW_CURSOR, GLFW_CURSOR_DISABLED);
	glfwSetCursorPosCallback(window.getWindow(), mouse_callback);
	glfwSetFramebufferSizeCallback(window.getWindow(), framebuffer_size_callback);
	//==================

	//==================
	//Create our 2D quad
	Shader quadShader("QuadShader.vert", "QuadShader.frag");
	quadShader.use();

	float vertices[] = {
		// positions         // texture coords
		 1.0f,  1.0f, 0.0f,  0.0f, 0.0f,   // top right
		 1.0f, -1.0f, 0.0f,  0.0f, 1.0f,   // bottom right
		-1.0f, -1.0f, 0.0f,  1.0f, 1.0f,   // bottom left
		-1.0f,  1.0f, 0.0f,  1.0f, 0.0f    // top left 
	};
	unsigned int indices[] = {
		0, 1, 3,  // first Triangle
		1, 2, 3   // second Triangle
	};

	//send to the GPU
	VertexArray VAO;
	VertexBuffer VBO(vertices, sizeof(vertices));
	ElementBuffer EBO(indices, sizeof(indices));

	VertexBufferLayout layout;
	layout.push<float>(3);
	layout.push<float>(2);
	VAO.addBuffer(VBO, layout);
	//==================

	//==================
	//create quad texture
	Texture quadTexture(nullptr, WIDTH, HEIGHT);

	//bind OpenGL array to CUDA
	hipArray_t texturePtr;
	hipGraphicsResource* cudaResource;
	hipGraphicsGLRegisterImage(&cudaResource, quadTexture.id(), GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
	hipGraphicsMapResources(1, &cudaResource, 0);
	hipGraphicsSubResourceGetMappedArray(&texturePtr, cudaResource, 0, 0);
	//==================


	//==================
	//Create CUDA Renderer
	Renderer renderer(WIDTH, HEIGHT);
	createScene(renderer);
	//=================


	//==================
	//Render our quad
	double lastTime = glfwGetTime();
	while (!window.close()) {
		processInput(window.getWindow());
		float currentFrame = (float)glfwGetTime();
		deltaTime = currentFrame - lastFrame;
		lastFrame = currentFrame;

		window.clear();

		updateTexture(renderer, &texturePtr);

		VAO.bind();
		quadTexture.bind();
		glDrawElements(GL_TRIANGLES, sizeof(indices), GL_UNSIGNED_INT, 0);
		quadTexture.unbind();
		VAO.unbind();

		window.swapBuffers();
		window.getEvents();
	}
	return 0;
}

bool lockFPS(uint32_t FPS) {
	static double lastTime = glfwGetTime();
	if (!(glfwGetTime() < lastTime + (1.0 / FPS))) {
		lastTime += (1.0 / FPS);
		return true;
	}
	return false;
}

void processInput(GLFWwindow* window) {
	if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
		glfwSetWindowShouldClose(window, true);
	if (glfwGetKey(window, GLFW_KEY_M) == GLFW_PRESS) {
		if (glfwGetInputMode(window, GLFW_CURSOR) == GLFW_CURSOR_DISABLED) {
			glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_NORMAL);
		}
		else {
			glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
		}
	}

	bool hasInput = false;
	Camera_Movement movement;
	if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS) {
		movement = FORWARD;
		hasInput = true;
	}
	if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS) {
		movement = BACKWARD;
		hasInput = true;
	}
	if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS) {
		movement = RIGHT;
		hasInput = true;
	}
	if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS) {
		movement = LEFT;
		hasInput = true;
	}
	if (glfwGetKey(window, GLFW_KEY_Q) == GLFW_PRESS) {
		movement = DOWN;
		hasInput = true;
	}
	if (glfwGetKey(window, GLFW_KEY_E) == GLFW_PRESS) {
		movement = UP;
		hasInput = true;
	}
	if (glfwGetKey(window, GLFW_KEY_L) == GLFW_PRESS) {
		globalLight = true;
	}
	if (glfwGetKey(window, GLFW_KEY_K) == GLFW_PRESS) {
		globalLight = false;
	}

	if (hasInput) {
		camera.ProcessKeyboard(movement, deltaTime);
		samples = SAMPLE_MIN;
	}

}
void framebuffer_size_callback(GLFWwindow* window, int width, int height) {
	glViewport(0, 0, width, height);
}

void mouse_callback(GLFWwindow* window, double xpos, double ypos) {
	if (firstMouse) {
		lastX = (float)xpos;
		lastY = (float)ypos;
		firstMouse = false;
	}

	float xoffset = (float)xpos - lastX;
	float yoffset = lastY - (float)ypos;

	lastX = (float)xpos;
	lastY = (float)ypos;

	camera.ProcessMouseMovement(xoffset, yoffset);
	samples = SAMPLE_MIN;
}

float randFloat(float a, float b) {
	float random = ((float)rand()) / (float)RAND_MAX;
	float diff = b - a;
	float r = random * diff;
	return a + r;
}

void createScene(Renderer& renderer) {
	int temp = 3;
	for (int a = -temp; a < temp; a++) {
		for (int b = -temp; b < temp; b++) {
			float choose_mat = randFloat(0, 1.1f);
			vec3 center(a + randFloat(0, 1.0f), 0.201f, b + randFloat(0, 1.0f));
			glm::vec3 color = glm::vec3(randFloat(0.1f, 1.0f), randFloat(0.1f, 1.0f), randFloat(0.1f, 1.0f));
			if (choose_mat < 0.5f) {
				Material diffuse(color);
				renderer.addSphere(Sphere(center, 0.2f), diffuse);
			} else if (choose_mat < 1.0f) {
				Material metal(color, 1, randFloat(0.0f, 0.3f));
				renderer.addSphere(Sphere(center, 0.2f), metal);
			} else {
				Material emissive(color, 0, 0, 0, 0, 1);
				renderer.addSphere(Sphere(center, 0.2f), emissive);
			}
		}
	}
	Material dielect(glm::vec3(0), 0, 0, 1, 1.5f);
	renderer.addSphere(Sphere(glm::vec3(0, 1, -0.5f), 0.5), dielect);

	Material planeMat(vec3(0.6f),1,0.1f);
	renderer.addPlane(Plane(vec3(0.0f, 0.0f, -5.0f), vec3(0, 1, 0)), planeMat);

	renderer.addLight(Light(vec3(0),1.0f));

	vec3 scale(0.2f);
	{
		Model tree("Objects/tree.obj");
		glm::mat4 matrix(1.0f);
		matrix = glm::translate(matrix, glm::vec3(-4.0f, 0, 0));
		matrix = glm::scale(matrix, scale);
		tree.transform(matrix);
		renderer.addModel(tree, Material(vec3(0,0,1.0f)));
	}
	{
		Model tree("Objects/tree.obj");
		glm::mat4 matrix(1.0f);
		matrix = glm::translate(matrix, glm::vec3(0.0f,0.0f, -4.0f));
		matrix = glm::scale(matrix, scale);
		tree.transform(matrix);
		renderer.addModel(tree, Material(vec3(0,1.0f,0)));
	}
	{
		Model tree("Objects/tree.obj");
		glm::mat4 matrix(1.0f);
		matrix = glm::translate(matrix, glm::vec3(4.0f, 0, 0));
		matrix = glm::scale(matrix, scale);
		tree.transform(matrix);
		renderer.addModel(tree, Material(vec3(1.0f,0,0)));
	}
}

void updateTexture(Renderer& renderer, hipArray_t* writeTo) {
	struct hipResourceDesc description;
	memset(&description, 0, sizeof(description));
	description.resType = hipResourceTypeArray;
	description.res.array.array = *writeTo;

	hipSurfaceObject_t write;
	hipCreateSurfaceObject(&write, &description);
	renderer.render(write, camera, QUASI_SAMPLE_N, globalLight);
}